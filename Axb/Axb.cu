
#include <stdio.h>
#include <fstream>
#include <iomanip>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <assert.h>

#include "hip/hip_runtime.h" 


#include "hipblas.h"

#define prec_save 10

#define BLOCKSIZE 256

#define BLOCKSIZEX 16
#define BLOCKSIZEY 16

/**********************/
/* HANDLE CUDA ERRORS */
/**********************/
#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

/************************/
/* HANDLE cublas ERRORS */
/************************/

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)

/************************************/
/* SAVE REAL ARRAY FROM CPU TO FILE */
/************************************/
template <class T>
void saveCPUrealtxt(const T * h_in, const char *filename, const int M) {

    std::ofstream outfile;
    outfile.open(filename);
    for (int i = 0; i < M; i++) outfile << std::setprecision(prec_save) << h_in[i] << "\n";
    outfile.close();

}
/************************************/
/* SAVE REAL ARRAY FROM GPU TO FILE */
/************************************/
template <class T>
void saveGPUrealtxt(const T * d_in, const char *filename, const int M) {

    T *h_in = (T *)malloc(M * sizeof(T));

    cudacall(hipMemcpy(h_in, d_in, M * sizeof(T), hipMemcpyDeviceToHost));

    std::ofstream outfile;
    outfile.open(filename);
    for (int i = 0; i < M; i++) outfile << std::setprecision(prec_save) << h_in[i] << "\n";
    outfile.close();

}

/***************************************************/
/* FUNCTION TO SET THE VALUES OF THE HANKEL MATRIX */
/***************************************************/
// --- https://en.wikipedia.org/wiki/Hankel_matrix
void setHankelMatrix(double * __restrict h_A, const int N) {

    double *h_atemp = (double *)malloc((2 * N - 1) * sizeof(double));

    // --- Initialize random seed
    srand(time(NULL));

    // --- Generate random numbers
    for (int k = 0; k < 2 * N - 1; k++) h_atemp[k] = rand();

    // --- Fill the Hankel matrix. The Hankel matrix is symmetric, so filling by row or column is equivalent.
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            h_A[i * N + j] = h_atemp[(i + 1) + (j + 1) - 2];

    free(h_atemp);

}

/***********************************************/
/* FUNCTION TO COMPUTE THE COEFFICIENTS VECTOR */
/***********************************************/
void computeCoefficientsVector(const double * __restrict h_A, const double * __restrict h_xref, 
                               double * __restrict h_y, const int N) {

    for (int k = 0; k < N; k++) h_y[k] = 0.f;

    for (int m = 0; m < N; m++)
        for (int n = 0; n < N; n++)
            h_y[m] = h_y[m] + h_A[n * N + m] * h_xref[n];

}


/************************************/
/* COEFFICIENT REARRANGING FUNCTION */
/************************************/
void rearrange(double *vec, int *pivotArray, int N){
    for (int i = 0; i < N; i++) {
        double temp = vec[i];
        vec[i] = vec[pivotArray[i] - 1];
        vec[pivotArray[i] - 1] = temp;
    }   
}

/********/
/* MAIN */
/********/
int main() {

    const unsigned int N = 1500;

    const unsigned int Nmatrices = 1;

    // --- CUBLAS initialization
    hipblasHandle_t cublas_handle;
    cublascall(hipblasCreate(&cublas_handle));

    hipEvent_t startLU, startApr1, startApr2;
    hipEvent_t stopLU, stopApr1, stopApr2;
    hipEventCreate(&startLU);
    hipEventCreate(&startApr1);
    hipEventCreate(&startApr2);
    hipEventCreate(&stopLU);
    hipEventCreate(&stopApr1);
    hipEventCreate(&stopApr2);

    float timingLU=0;
    float timingApr1=0;
    float timingApr2=0;

    /***********************/
    /* SETTING THE PROBLEM */
    /***********************/
    // --- Matrices to be inverted (only one in this example)
    double *h_A = (double *)malloc(N * N * Nmatrices * sizeof(double));

    // --- Setting the Hankel matrix
    setHankelMatrix(h_A, N);

    // --- Defining the solution
    double *h_xref = (double *)malloc(N * sizeof(double));
    for (int k = 0; k < N; k++) h_xref[k] = 1.f;

    // --- Coefficient vectors (only one in this example)
    double *h_y = (double *)malloc(N * sizeof(double));

    computeCoefficientsVector(h_A, h_xref, h_y, N);

    // --- Result (only one in this example)
    double *h_x = (double *)malloc(N * sizeof(double));

    // --- Allocate device space for the input matrices 
    double *d_A; cudacall(hipMalloc(&d_A, N * N * Nmatrices * sizeof(double)));
    double *d_y; cudacall(hipMalloc(&d_y, N *                 sizeof(double)));
    double *d_x; cudacall(hipMalloc(&d_x, N *                 sizeof(double)));

    // --- Move the relevant matrices from host to device
    cudacall(hipMemcpy(d_A, h_A, N * N * Nmatrices * sizeof(double), hipMemcpyHostToDevice));
    cudacall(hipMemcpy(d_y, h_y, N *                 sizeof(double), hipMemcpyHostToDevice));

    /**********************************/
    /* COMPUTING THE LU DECOMPOSITION */
    /**********************************/
    hipEventRecord(startLU, 0);

    // --- Creating the array of pointers needed as input/output to the batched getrf
    double **h_inout_pointers = (double **)malloc(Nmatrices * sizeof(double *));
    for (int i = 0; i < Nmatrices; i++) h_inout_pointers[i] = d_A + i * N * N;

    double **d_inout_pointers;
    cudacall(hipMalloc(&d_inout_pointers, Nmatrices * sizeof(double *)));
    cudacall(hipMemcpy(d_inout_pointers, h_inout_pointers, Nmatrices * sizeof(double *), hipMemcpyHostToDevice));
    free(h_inout_pointers);

    int *d_pivotArray; cudacall(hipMalloc(&d_pivotArray, N * Nmatrices * sizeof(int)));
    int *d_InfoArray;  cudacall(hipMalloc(&d_InfoArray,      Nmatrices * sizeof(int)));

    int *h_InfoArray  = (int *)malloc(Nmatrices * sizeof(int));

    cublascall(hipblasDgetrfBatched(cublas_handle, N, d_inout_pointers, N, d_pivotArray, d_InfoArray, Nmatrices));
    //cublascall(cublasDgetrfBatched(cublas_handle, N, d_inout_pointers, N, NULL, d_InfoArray, Nmatrices));

    cudacall(hipMemcpy(h_InfoArray, d_InfoArray, Nmatrices * sizeof(int), hipMemcpyDeviceToHost));

    for (int i = 0; i < Nmatrices; i++)
        if (h_InfoArray[i] != 0) {
            fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
            hipDeviceReset();
            exit(EXIT_FAILURE);
        }

    hipEventRecord(stopLU, 0);
    hipEventSynchronize(stopLU);

    hipEventElapsedTime(&timingLU, startLU, stopLU);

    hipEventDestroy(startLU);
    hipEventDestroy(stopLU);

    printf("The elapsed time for LU decomp in gpu was %.2f ms\n", timingLU);

    /*********************************/
    /* CHECKING THE LU DECOMPOSITION */
    /*********************************/
    saveCPUrealtxt(h_A,          "output/A.txt", N * N);
    saveCPUrealtxt(h_y,          "output/y.txt", N);
    saveGPUrealtxt(d_A,          "output/Adecomposed.txt", N * N);
    saveGPUrealtxt(d_pivotArray, "output/pivotArray.txt", N);

    /******************************************************************************/
    /* APPROACH NR.1: COMPUTE THE INVERSE OF A STARTING FROM ITS LU DECOMPOSITION */
    /******************************************************************************/
    hipEventRecord(startApr1, 0);

    // --- Allocate device space for the inverted matrices 
    double *d_Ainv; cudacall(hipMalloc(&d_Ainv, N * N * Nmatrices * sizeof(double)));

    // --- Creating the array of pointers needed as output to the batched getri
    double **h_out_pointers = (double **)malloc(Nmatrices * sizeof(double *));
    for (int i = 0; i < Nmatrices; i++) h_out_pointers[i] = (double *)((char*)d_Ainv + i * ((size_t)N * N) * sizeof(double));

    double **d_out_pointers;
    cudacall(hipMalloc(&d_out_pointers, Nmatrices*sizeof(double *)));
    cudacall(hipMemcpy(d_out_pointers, h_out_pointers, Nmatrices*sizeof(double *), hipMemcpyHostToDevice));
    free(h_out_pointers);

    cublascall(hipblasDgetriBatched(cublas_handle, N, (const double **)d_inout_pointers, N, d_pivotArray, d_out_pointers, N, d_InfoArray, Nmatrices));

    cudacall(hipMemcpy(h_InfoArray, d_InfoArray, Nmatrices * sizeof(int), hipMemcpyDeviceToHost));

    for (int i = 0; i < Nmatrices; i++)
        if (h_InfoArray[i] != 0) {
        fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
        hipDeviceReset();
        exit(EXIT_FAILURE);
        }

    double alpha1 = 1.f;
    double beta1 = 0.f;

    cublascall(hipblasDgemv(cublas_handle, HIPBLAS_OP_N, N, N, &alpha1, d_Ainv, N, d_y, 1, &beta1, d_x, 1));

    hipEventRecord(stopApr1, 0);
    hipEventSynchronize(stopApr1);

    hipEventElapsedTime(&timingApr1, startApr1, stopApr1);

    hipEventDestroy(startApr1);
    hipEventDestroy(stopApr1);

    printf("The elapsed time for Approach 1 in gpu was %.2f ms\n", timingApr1);

    /**************************/
    /* CHECKING APPROACH NR.1 */
    /**************************/
    saveGPUrealtxt(d_x, "output/xApproach1.txt", N);

    /*************************************************************/
    /* APPROACH NR.2: INVERT UPPER AND LOWER TRIANGULAR MATRICES */
    /*************************************************************/
    hipEventRecord(startApr2, 0);

    double *d_P; cudacall(hipMalloc(&d_P, N * N * sizeof(double)));

    cudacall(hipMemcpy(h_y, d_y, N * Nmatrices * sizeof(int), hipMemcpyDeviceToHost));
    int *h_pivotArray = (int *)malloc(N * Nmatrices*sizeof(int));
    cudacall(hipMemcpy(h_pivotArray, d_pivotArray, N * Nmatrices * sizeof(int), hipMemcpyDeviceToHost));

    rearrange(h_y, h_pivotArray, N);
    cudacall(hipMemcpy(d_y, h_y, N * Nmatrices * sizeof(double), hipMemcpyHostToDevice));

    // --- Now P*A=L*U
    //     Linear system A*x=y => P.'*L*U*x=y => L*U*x=P*y

    // --- 1st phase - solve Ly = b 
    const double alpha = 1.f;

    // --- Function solves the triangular linear system with multiple right hand sides, function overrides b as a result 

    // --- Lower triangular part
    cublascall(hipblasDtrsm(cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, 1, &alpha, d_A, N, d_y, N));

    // --- Upper triangular part
    cublascall(hipblasDtrsm(cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, N, 1, &alpha, d_A, N, d_y, N));

    hipEventRecord(stopApr2, 0);
    hipEventSynchronize(stopApr2);

    hipEventElapsedTime(&timingApr2, startApr2, stopApr2);

    hipEventDestroy(startApr2);
    hipEventDestroy(stopApr2);

    printf("The elapsed time for Approach 2 in gpu was %.2f ms\n", timingLU + timingApr2);
    
    /**************************/
    /* CHECKING APPROACH NR.2 */
    /**************************/
    saveGPUrealtxt(d_y, "output/xApproach2.txt", N);

    return 0;
}
