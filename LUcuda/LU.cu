
/*
 Thanks to
 https://stackoverflow.com/questions/22814040/lu-decomposition-in-cuda
*/

#include <stdio.h>

#include "hip/hip_runtime.h" 


#include "hipblas.h"

/**********************/ 
/* HANDLE CUDA ERRORS */  
/**********************/
#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)
    
/************************/
/* HANDLE cublas ERRORS */
/************************/

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)



int main() {

    const unsigned int Nmatrices = 1;

    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    // --- Matrices to be inverted (only one in this example)
#if 1
    const unsigned int N = 3; 
    float *h_A = new float[N*N*Nmatrices];

    h_A[0] = 4.f;  
    h_A[1] = 3.f;
    h_A[2] = 8.f;
    h_A[3] = 9.f;
    h_A[4] = 5.f; 
    h_A[5] = 1.f; 
    h_A[6] = 2.f; 
    h_A[7] = 7.f;
    h_A[8] = 6.f;
#endif
#if 0
    const unsigned int N = 2; 
    float *h_A = new float[N*N*Nmatrices];
    h_A[0] = 3.f;
    h_A[1] = 4.f;
    h_A[2] = 1.f;
    h_A[3] = 2.f;
#endif
    // --- Allocate device matrices 
    float *d_A; cudacall(hipMalloc((void**)&d_A, N*N*Nmatrices*sizeof(float)));

    // --- Move the matrix to be inverted from host to device
    cudacall(hipMemcpy(d_A,h_A,N*N*Nmatrices*sizeof(float),hipMemcpyHostToDevice));

    // --- Creating the array of pointers needed as input to the batched getrf
    float **h_inout_pointers = (float **)malloc(Nmatrices*sizeof(float *));
    for (int i=0; i<Nmatrices; i++) h_inout_pointers[i]=(float *)((char*)d_A+i*((size_t)N*N)*sizeof(float));

    float **d_inout_pointers;
    cudacall(hipMalloc((void**)&d_inout_pointers, Nmatrices*sizeof(float *)));
    cudacall(hipMemcpy(d_inout_pointers,h_inout_pointers,Nmatrices*sizeof(float *),hipMemcpyHostToDevice));
    free(h_inout_pointers);

    int *d_PivotArray; cudacall(hipMalloc((void**)&d_PivotArray, N*Nmatrices*sizeof(int)));
    int *d_InfoArray;  cudacall(hipMalloc((void**)&d_InfoArray,  Nmatrices*sizeof(int)));

    int *h_PivotArray = (int *)malloc(N*Nmatrices*sizeof(int));
    int *h_InfoArray  = (int *)malloc(  Nmatrices*sizeof(int));

    cublascall(hipblasSgetrfBatched(handle, N, d_inout_pointers, N, d_PivotArray, d_InfoArray, Nmatrices));
    //cublasSafeCall(cublasSgetrfBatched(handle, N, d_inout_pointers, N, NULL, d_InfoArray, Nmatrices));

    cudacall(hipMemcpy(h_InfoArray,d_InfoArray,Nmatrices*sizeof(int),hipMemcpyDeviceToHost));

    for (int i = 0; i < Nmatrices; i++)
        if (h_InfoArray[i]  != 0) {
            fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
            hipDeviceReset();
            exit(EXIT_FAILURE);
        }

    cudacall(hipMemcpy(h_A,d_A,N*N*sizeof(float),hipMemcpyDeviceToHost));
    cudacall(hipMemcpy(h_PivotArray,d_PivotArray,N*Nmatrices*sizeof(int),hipMemcpyDeviceToHost));

    printf("\nCombined LU matrix in column major form: \n");
    for (int i=0; i<N*N; i++) printf("A[%i]=%f\n", i, h_A[i]);

    printf("\n Pivot Array\n");    
    for (int i=0; i<N; i++) printf("P[%i]=%i\n", i, h_PivotArray[i]);

    // Compute the determinant of h_A 
    float det = h_A[0];
    for(int i = 1; i < N; ++i)
      det *= h_A[i*(N+1)];

     printf("determinant of A: %f \n", det);

    return 0;
}

