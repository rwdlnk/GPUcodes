
#include <stdio.h>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)

/***********************************/
/* SAVE REAL ARRAY FROM GPU TO FILE */
/************************************/
template <class T>
void saveGPUrealtxt(const T * d_in, const char *filename, const int M) {

    T *h_in = (T *)malloc(M * sizeof(T));

    cudacall(hipMemcpy(h_in, d_in, M * sizeof(T), hipMemcpyDeviceToHost));

    std::ofstream outfile;
    outfile.open(filename);
    for (int i = 0; i < M; i++) outfile << std::setprecision(10) << h_in[i] << "\n";
    outfile.close();

}

int* cublas_lu(int m, int n, double* a)
{
    hipblasHandle_t handle;
    double **devPtrA = 0;
    double **devPtrA_dev = NULL;
    int *d_pivot_array;
    int *d_info_array;
    int rowsA = m;
    int colsA = n;
    int matrixSizeA;

    // allocate the pivoting vector and the info array
    cudacall(hipMalloc(&d_pivot_array, n * sizeof(int)));
    cudacall(hipMalloc(&d_info_array, sizeof(int)));

    cublascall(hipblasCreate(&handle));
    matrixSizeA = rowsA * colsA;

    devPtrA =(double **)malloc(1 * sizeof(*devPtrA));
    if (devPtrA == NULL){ 
     perror("malloc"); 
     exit(EXIT_FAILURE); 
    }
	
    cudacall(hipMalloc(devPtrA, matrixSizeA * sizeof(double)));
    cudacall(hipMalloc(&devPtrA_dev, 1 * sizeof(*devPtrA)));

    cudacall(hipMemcpy(devPtrA_dev, devPtrA, 1 * sizeof(*devPtrA), hipMemcpyHostToDevice));
    
    cublascall(hipblasSetMatrix(rowsA, colsA, sizeof(a[0]), a, rowsA, devPtrA[0], rowsA));

    // Perform LU decomposition
    cublascall(hipblasDgetrfBatched(handle, m, devPtrA_dev, m, d_pivot_array, d_info_array, 1));

    cublascall(hipblasGetMatrix(m, n, sizeof(double), devPtrA[0], m, a, m));

   saveGPUrealtxt(d_pivot_array, "pivot.txt", m);

   return d_pivot_array;
}


int main()
{
#if 0
	const int n = 4;
        // A in column major form.
	double A[n * n] = { 1.0, 1.0,  3.0, -2.0,
	  	            1.0, 2.0, -1.0,  3.0,
			    2.0, 1.0,  3.0, -1.0,
			    1.0, 2.0, -2.0,  1.0 };
#endif
#if 1
	const int n = 3;
        // A in column major form.
	double A[n * n] = { 4.0, 3.0, 8.0,
	  	            9.0, 5.0, 1.0,
			    2.0, 7.0, 6.0 };
#endif
#if 0
	const int n = 2;
        // A in column major form.
	double A[n * n] = { 4.0, 3.0,
			    1.0, 5.0 };
#endif
#if 0
	const int n = 2;
        // A in column major form.
	double A[n * n] = { 3.0, 4.0,
			    5.0, 1.0 };
#endif
    printf("Initial A matrix: \n");
    for(int j=0; j<n; j++)
    {
        for(int i=0; i<n; i++)
            fprintf(stdout,"%f\t",A[i*n+j]);
        fprintf(stdout,"\n");
    }					

    int * d_pivot;
    cudacall(hipMalloc(&d_pivot, n * sizeof(int)));
    int *h_pivot = (int *)malloc(n * sizeof(int));

    d_pivot = cublas_lu(n, n, A);

    cudacall(hipMemcpy(h_pivot, d_pivot, n*sizeof(int), hipMemcpyDeviceToHost));

    fprintf(stdout, "Combined LU matrix (pivot applied):\n");
    for(int j=0; j<n; j++)
    {
        for(int i=0; i<n; i++)
            fprintf(stdout,"%f\t",A[i*n+j]);
        fprintf(stdout,"\n");
    }					

   double det = A[0];
   for(int i = 1; i < n; ++i)
     det *= A[i*(n+1)];

   for(int i = 0; i < n; ++i){
     printf(" P[%d]: %d \n", i, h_pivot[i]-1);
     if(h_pivot[i]-1 != i)
       det *= -1.;
   }

   fprintf(stdout," det: %f \n", det);
}
